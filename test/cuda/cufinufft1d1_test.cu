#include "hip/hip_runtime.h"
#include <cmath>
#include <complex>
#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>
#include <random>

#include <cufinufft/utils.h>
#include <cufinufft_eitherprec.h>
using cufinufft::utils::infnorm;

int main(int argc, char *argv[]) {
    if (argc != 6) {
        fprintf(stderr, "Usage: cufinufft1d1_test method N M tol checktol\n"
                        "Arguments:\n"
                        "  method: One of\n"
                        "    1: nupts driven, or\n"
                        "    2: sub-problem\n"
                        "  N: The size of the 1D array\n"
                        "  M: The number of non-uniform points\n"
                        "  tol: NUFFT tolerance.\n"
                        "  checktol: relative error to pass test\n");
        return 1;
    }
    int method = atoi(argv[1]);
    int N = atof(argv[2]);
    int M = atof(argv[3]);
    CUFINUFFT_FLT tol = atof(argv[4]);
    CUFINUFFT_FLT checktol = atof(argv[5]);

    int iflag = 1;

    std::cout << std::scientific << std::setprecision(3);
    int ier;

    CUFINUFFT_FLT *x;
    CUFINUFFT_CPX *c, *fk;
    hipHostMalloc(&x, M * sizeof(CUFINUFFT_FLT));
    hipHostMalloc(&c, M * sizeof(CUFINUFFT_CPX));
    hipHostMalloc(&fk, N * sizeof(CUFINUFFT_CPX));

    CUFINUFFT_FLT *d_x;
    CUCPX *d_c, *d_fk;
    checkCudaErrors(hipMalloc(&d_x, M * sizeof(CUFINUFFT_FLT)));
    checkCudaErrors(hipMalloc(&d_c, M * sizeof(CUCPX)));
    checkCudaErrors(hipMalloc(&d_fk, N * sizeof(CUCPX)));

    std::default_random_engine eng(1);
    std::uniform_real_distribution<CUFINUFFT_FLT> dist11(-1, 1);
    auto randm11 = [&eng, &dist11]() { return dist11(eng); };

    // Making data
    for (int i = 0; i < M; i++) {
        x[i] = M_PI * randm11(); // x in [-pi,pi)
        c[i].real(randm11());
        c[i].imag(randm11());
    }

    checkCudaErrors(hipMemcpy(d_x, x, M * sizeof(CUFINUFFT_FLT), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_c, c, M * sizeof(CUFINUFFT_CPX), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    float milliseconds = 0;
    float totaltime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // warm up CUFFT (is slow, takes around 0.2 sec... )
    hipEventRecord(start);
    {
        int nf1 = 1;
        hipfftHandle fftplan;
        hipfftPlan1d(&fftplan, nf1, CUFFT_TYPE, 1);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] dummy warmup call to CUFFT\t %.3g s\n", milliseconds / 1000);

    // now to our tests...
    CUFINUFFT_PLAN dplan;
    int dim = 1;
    int type = 1;

    // Here we setup our own opts, for gpu_method.
    cufinufft_opts opts;
    ier = CUFINUFFT_DEFAULT_OPTS(type, dim, &opts);
    if (ier != 0) {
        printf("err %d: CUFINUFFT_DEFAULT_OPTS\n", ier);
        return ier;
    }

    opts.gpu_method = method;

    int nmodes[3];
    int ntransf = 1;
    int maxbatchsize = 1;
    nmodes[0] = N;
    nmodes[1] = 1;
    nmodes[2] = 1;
    hipEventRecord(start);
    ier = CUFINUFFT_MAKEPLAN(type, dim, nmodes, iflag, ntransf, tol, maxbatchsize, &dplan, &opts);
    if (ier != 0) {
        printf("err: cufinufft1d_plan\n");
        return ier;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    totaltime += milliseconds;
    printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds / 1000);

    hipEventRecord(start);
    ier = CUFINUFFT_SETPTS(M, d_x, NULL, NULL, 0, NULL, NULL, NULL, dplan);
    if (ier != 0) {
        printf("err: cufinufft_setpts\n");
        return ier;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    totaltime += milliseconds;
    printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds / 1000);

    hipEventRecord(start);
    ier = CUFINUFFT_EXECUTE(d_c, d_fk, dplan);
    if (ier != 0) {
        printf("err: cufinufft1d1_exec\n");
        return ier;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    totaltime += milliseconds;
    float exec_ms = milliseconds;
    printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds / 1000);

    hipEventRecord(start);
    ier = CUFINUFFT_DESTROY(dplan);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    totaltime += milliseconds;
    printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds / 1000);

    checkCudaErrors(hipMemcpy(fk, d_fk, N * sizeof(CUCPX), hipMemcpyDeviceToHost));

    printf("[Method %d] %d NU pts to %d U pts in %.3g s:      %.3g NU pts/s\n", opts.gpu_method, M, N,
           totaltime / 1000, M / totaltime * 1000);
    printf("\t\t\t\t\t(exec-only thoughput: %.3g NU pts/s)\n", M / exec_ms * 1000);

    int nt1 = (int)(0.37 * N); // choose some mode index to check
    CUFINUFFT_CPX Ft = CUFINUFFT_CPX(0, 0), J = IMA * (CUFINUFFT_FLT)iflag;
    for (int j = 0; j < M; ++j)
        Ft += c[j] * exp(J * (nt1 * x[j])); // crude direct
    int it = N / 2 + nt1;                  // index in complex F as 1d array

    CUFINUFFT_FLT rel_error = abs(Ft - fk[it]) / infnorm(N, fk);
    printf("[gpu   ] one mode: rel err in F[%ld] is %.3g\n", (long)nt1, rel_error);

    hipHostFree(x);
    hipHostFree(c);
    hipHostFree(fk);
    hipFree(d_x);
    hipFree(d_c);
    hipFree(d_fk);

    return rel_error > checktol;
}
