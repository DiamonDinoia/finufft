#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>

#include "cufinufft/common.h"
#include "cufinufft/contrib/common.h"
#include "cufinufft/precision_independent.h"
#include <hip/hip_complex.h>

/* Kernel for computing approximations of exact Fourier series coeffs of
   cnufftspread's real symmetric kernel. */
// a , f are intermediate results from function onedim_fseries_kernel_precomp()
// (see cufinufft/contrib/common.cpp for description)
__global__ void FseriesKernelCompute(int nf1, int nf2, int nf3, CUFINUFFT_FLT *f, hipDoubleComplex *a,
                                     CUFINUFFT_FLT *fwkerhalf1, CUFINUFFT_FLT *fwkerhalf2, CUFINUFFT_FLT *fwkerhalf3,
                                     int ns) {
    CUFINUFFT_FLT J2 = ns / 2.0;
    int q = (int)(2 + 3.0 * J2);
    int nf;
    hipDoubleComplex *at = a + threadIdx.y * MAX_NQUAD;
    CUFINUFFT_FLT *ft = f + threadIdx.y * MAX_NQUAD;
    CUFINUFFT_FLT *oarr;
    if (threadIdx.y == 0) {
        oarr = fwkerhalf1;
        nf = nf1;
    } else if (threadIdx.y == 1) {
        oarr = fwkerhalf2;
        nf = nf2;
    } else {
        oarr = fwkerhalf3;
        nf = nf3;
    }

    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nf / 2 + 1; i += blockDim.x * gridDim.x) {
        int brk = 0.5 + i;
        CUFINUFFT_FLT x = 0.0;
        for (int n = 0; n < q; n++) {
            x += ft[n] * 2 * (pow(cabs(at[n]), brk) * cos(brk * carg(at[n])));
        }
        oarr[i] = x;
    }
}

int CUFSERIESKERNELCOMPUTE(int dim, int nf1, int nf2, int nf3, CUFINUFFT_FLT *d_f, hipDoubleComplex *d_a,
                           CUFINUFFT_FLT *d_fwkerhalf1, CUFINUFFT_FLT *d_fwkerhalf2, CUFINUFFT_FLT *d_fwkerhalf3,
                           int ns)
/*
    wrapper for approximation of Fourier series of real symmetric spreading
    kernel.

    Melody Shih 2/20/22
*/
{
    int nout = max(max(nf1 / 2 + 1, nf2 / 2 + 1), nf3 / 2 + 1);

    dim3 threadsPerBlock(16, dim);
    dim3 numBlocks((nout + 16 - 1) / 16, 1);

    FseriesKernelCompute<<<numBlocks, threadsPerBlock>>>(nf1, nf2, nf3, d_f, d_a, d_fwkerhalf1, d_fwkerhalf2,
                                                         d_fwkerhalf3, ns);
    return 0;
}
